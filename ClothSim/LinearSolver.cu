#include "hip/hip_runtime.h"
#include "LinearSolver.h"

namespace cloth
{
	template <>
	struct CusparseCaller<double>
	{
		static void createCsr(hipsparseSpMatDescr_t* mat, int n, int nnz, int* rowPtr, int* colIdx, double* values)
		{
			checkCudaErrors(hipsparseCreateCsr(
				mat, n, n, nnz, rowPtr, colIdx, values,
				HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
		}

		static void createDnVec(hipsparseDnVecDescr_t* vec, int n, double* values)
		{
			checkCudaErrors(hipsparseCreateDnVec(vec, n, values, HIP_R_64F));
		}

		static void mv_bufferSize(hipsparseHandle_t handle, const double* alpha, hipsparseSpMatDescr_t matA, hipsparseDnVecDescr_t vecAx,
			const double* beta, hipsparseDnVecDescr_t vecx, size_t* bufferSize)
		{
			checkCudaErrors(hipsparseSpMV_bufferSize(
				handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecAx, beta, vecx,
				HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, bufferSize));
		}

		static void mv(hipsparseHandle_t handle, const double* alpha, hipsparseSpMatDescr_t matA, hipsparseDnVecDescr_t vecx,
			const double* beta, hipsparseDnVecDescr_t vecAx, void* buffer)
		{
			checkCudaErrors(hipsparseSpMV(
				handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecx, beta, vecAx,
				HIP_R_64F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
		}

		static void ilu_bufferSize(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, double* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, int* bufferSize)
		{
			checkCudaErrors(hipsparseDcsrilu02_bufferSize(handle, n, nnz, descr, values, rowPtr, colIdx, info, bufferSize));
		}

		static void ilu_analysis(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, const double* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseDcsrilu02_analysis(handle, n, nnz, descr, values, rowPtr, colIdx, info,
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void ilu(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, double* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseDcsrilu02(handle, n, nnz, descr, values, rowPtr, colIdx, info,
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void sv2_bufferSize(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, double* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, int* bufferSize)
		{
			checkCudaErrors(hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				n, nnz, descr, values, rowPtr, colIdx, info, bufferSize));
		}

		static void sv2_analysis(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, const double* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz,
				descr, values, rowPtr, colIdx, info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void sv2_solve(hipsparseHandle_t handle, int n, int nnz, const double* alpha, const hipsparseMatDescr_t descr, const double* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, const double* x, double* y, void* buffer)
		{
			checkCudaErrors(hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, alpha,
				descr, values, rowPtr, colIdx, info, x, y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}
	};

	template <>
	struct CusparseCaller<float>
	{
		static void createCsr(hipsparseSpMatDescr_t* mat, int n, int nnz, int* rowPtr, int* colIdx, float* values)
		{
			checkCudaErrors(hipsparseCreateCsr(
				mat, n, n, nnz, rowPtr, colIdx, values,
				HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
		}

		static void createDnVec(hipsparseDnVecDescr_t* vec, int n, float* values)
		{
			checkCudaErrors(hipsparseCreateDnVec(vec, n, values, HIP_R_32F));
		}

		static void mv_bufferSize(hipsparseHandle_t handle, const float* alpha, hipsparseSpMatDescr_t matA, hipsparseDnVecDescr_t vecAx,
			const float* beta, hipsparseDnVecDescr_t vecx, size_t* bufferSize)
		{
			checkCudaErrors(hipsparseSpMV_bufferSize(
				handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecAx, &beta, vecx,
				HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, bufferSize));
		}

		static void mv(hipsparseHandle_t handle, const float* alpha, hipsparseSpMatDescr_t matA, hipsparseDnVecDescr_t vecx,
			const float* beta, hipsparseDnVecDescr_t vecAx, void* buffer)
		{
			checkCudaErrors(hipsparseSpMV(
				handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA,
				vecx, beta, vecAx, HIP_R_32F, HIPSPARSE_MV_ALG_DEFAULT, buffer));
		}

		static void ilu_bufferSize(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, float* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, int* bufferSize)
		{
			checkCudaErrors(hipsparseScsrilu02_bufferSize(handle, n, nnz, descr, values, rowPtr, colIdx, info, bufferSize));
		}

		static void ilu_analysis(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, const float* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseScsrilu02_analysis(handle, n, nnz, descr, values, rowPtr, colIdx, info,
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void ilu(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, float* values,
			const int* rowPtr, const int* colIdx, csrilu02Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseScsrilu02(handle, n, nnz, descr, values, rowPtr, colIdx, info,
				HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void sv2_bufferSize(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, float* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, int* bufferSize)
		{
			checkCudaErrors(hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
				n, nnz, descr, values, rowPtr, colIdx, info, bufferSize));
		}

		static void sv2_analysis(hipsparseHandle_t handle, int n, int nnz, const hipsparseMatDescr_t descr, const float* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, void* buffer)
		{
			checkCudaErrors(hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz,
				descr, values, rowPtr, colIdx, info, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}

		static void sv2_solve(hipsparseHandle_t handle, int n, int nnz, const float* alpha, const hipsparseMatDescr_t descr, const float* values,
			const int* rowPtr, const int* colIdx, csrsv2Info_t info, const float* x, float* y, void* buffer)
		{
			checkCudaErrors(hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, n, nnz, alpha,
				descr, values, rowPtr, colIdx, info, x, y, HIPSPARSE_SOLVE_POLICY_USE_LEVEL, buffer));
		}
	};

	template <>
	struct CusolverCaller<double>
	{
		static void cholesky(hipsolverSpHandle_t handle, int n, int nnz, hipsparseMatDescr_t descrA, const double* values, const int* rowPtr, const int* colIdx, const double* b, double* x, int* singularity)
		{
			hipsolverSpDcsrlsvchol(handle, n, nnz, descrA, values, rowPtr, colIdx, b, EPS, 2, x, singularity);  // symamd
			//Eigen::VecXx test(nnz);
			//hipMemcpy(test.data(), values, nnz * sizeof(double), hipMemcpyDeviceToHost);
			//std::cout << test << "\n\n\n";
			//test.resize(n);
			//hipMemcpy(test.data(), b, n * sizeof(double), hipMemcpyDeviceToHost);
			//std::cout << test;
		}
	};

	template <>
	struct CusolverCaller<float>
	{
		static void cholesky(hipsolverSpHandle_t handle, int n, int nnz, hipsparseMatDescr_t descrA, const float* values, const int* rowPtr, const int* colIdx, const float* b, float* x, int* singularity)
		{
			hipsolverSpScsrlsvchol(handle, n, nnz, descrA, values, rowPtr, colIdx, b, EPS, 2, x, singularity);  // symamd
		}
	};

	CudaMvWrapper::CudaMvWrapper(): 
		d_buffer(NULL), m_handle(NULL), m_matA(NULL), m_vecx(NULL), m_vecAx(NULL)
	{}

	CudaMvWrapper::~CudaMvWrapper()
	{
		if (d_buffer) hipFree(d_buffer);

		if (m_handle) hipsparseDestroy(m_handle);

		if (m_matA) hipsparseDestroySpMat(m_matA);
		if (m_vecx) hipsparseDestroyDnVec(m_vecx);
		if (m_vecAx) hipsparseDestroyDnVec(m_vecAx);
	}

	void CudaMvWrapper::initialize(SparseMatrix* A, Scalar* x, Scalar* Ax)
	{
		hipsparseCreate(&m_handle);

		CusparseCaller<Scalar>::createCsr(&m_matA, A->getn(), A->getnnz(), A->getRowPtr(), A->getColIdx(), A->getValue());
		CusparseCaller<Scalar>::createDnVec(&m_vecx, A->getn(), x);
		CusparseCaller<Scalar>::createDnVec(&m_vecAx, A->getn(), Ax);

		size_t buffer_size = 0;
		Scalar one = 1, zero = 0;
		CusparseCaller<Scalar>::mv_bufferSize(m_handle, &one, m_matA, m_vecAx, &zero, m_vecx, &buffer_size);

		hipMalloc((void**)&d_buffer, buffer_size);
	}

	void CudaMvWrapper::mv()
	{
		Scalar one = 1, zero = 0;
		CusparseCaller<Scalar>::mv(m_handle, &one, m_matA, m_vecx, &zero, m_vecAx, d_buffer);
	}

	LinearSolver::LinearSolver():
		d_r(NULL), d_p(NULL), d_z(NULL), d_Ap(NULL), m_precond(NULL)
	{}

	LinearSolver::~LinearSolver()
	{
		if (d_r) hipFree(d_r);
		if (d_p) hipFree(d_p);
		if (d_z) hipFree(d_z);
		if (d_Ap) hipFree(d_Ap);

		if (m_cublasHandle) hipblasDestroy(m_cublasHandle);
		if (m_cusolverSpHandle) hipsolverSpDestroy(m_cusolverSpHandle);
		if (m_descrA) hipsparseDestroyMatDescr(m_descrA);

		if (m_precond) delete m_precond;
	}

	void LinearSolver::initialize(SparseMatrix* mat, PrecondT pt)
	{
		m_matrix = mat;
		m_n = mat->getn();

		switch (pt)
		{
		case LinearSolver::NoPreconditionner:
			m_precond = new DummyPreconditioner(m_n);
			break;
		case LinearSolver::Diagnal:
			m_precond = new DiagnalPreconditioner(mat);
			break;
		case LinearSolver::Factorization:
			m_precond = new FactorizationPreconditioner(mat);
			break;
		}

		hipblasCreate(&m_cublasHandle);
		hipsolverSpCreate(&m_cusolverSpHandle);

		hipsparseCreateMatDescr(&m_descrA);
		hipsparseSetMatType(m_descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(m_descrA, HIPSPARSE_INDEX_BASE_ZERO);

		hipMalloc((void**)&d_r, m_n * sizeof(Scalar));
		hipMalloc((void**)&d_p, m_n * sizeof(Scalar));
		hipMalloc((void**)&d_z, m_n * sizeof(Scalar));
		hipMalloc((void**)&d_Ap, m_n * sizeof(Scalar));

		m_mv_caller.initialize(mat, d_p, d_Ap);
	}

	bool LinearSolver::cholesky(const Scalar* b, Scalar* x)
	{
		int singularity;
		CusolverCaller<Scalar>::cholesky(m_cusolverSpHandle, m_n, m_matrix->getnnz(), m_descrA, m_matrix->getValue(),
			m_matrix->getRowPtr(), m_matrix->getColIdx(), b, x, &singularity);
		return singularity < 0;
	}

	bool LinearSolver::conjugateGradient(const Scalar* b, Scalar* x)
	{
		Scalar one = 1.0, neg_one = -1.0;
		Scalar res, bnorm, alpha, beta;
		Scalar pAp, rz, old_rz;

		int nnz = m_matrix->getnnz();
		Scalar* values = m_matrix->getValue();
		const int* rowPtr = m_matrix->getRowPtr();
		const int* colIdx = m_matrix->getColIdx();

		// Perform analysis for ILU
		bool status = m_precond->analysis();
		if (!status)
		{
			//std::cerr << "Preconditioner analysis failed. EXIT." << std::endl;
			exit(-1);
		}

		// r0 = b - Ax
		CublasCaller<Scalar>::copy(m_cublasHandle, m_n, x, d_p);
		CublasCaller<Scalar>::copy(m_cublasHandle, m_n, b, d_r);

		m_mv_caller.mv();
		CublasCaller<Scalar>::axpy(m_cublasHandle, m_n, &neg_one, d_Ap, d_r);

		CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_r, d_r, &res);
		CublasCaller<Scalar>::dot(m_cublasHandle, m_n, b, b, &bnorm);

		//if (res / bnorm < eps) return;

		m_precond->solve(d_r, d_z);
		CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_z, d_p);
		CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_r, d_z, &rz);

		Eigen::VecXx test(m_n);

		int k = 0;
		for (k; k < m_n; ++k)
		{
			m_mv_caller.mv();
			CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_p, d_Ap, &pAp);
			alpha = rz / pAp;

			CublasCaller<Scalar>::axpy(m_cublasHandle, m_n, &alpha, d_p, x);
			alpha = -alpha;
			CublasCaller<Scalar>::axpy(m_cublasHandle, m_n, &alpha, d_Ap, d_r);
			old_rz = rz;

			CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_r, d_r, &res);
			//std::cout << "\t iter: " << k << " rTr: " << res << std::endl;

			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_p, d_Ap);
			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, x, d_p);
			//CusparseCaller<Scalar>::mv(m_cusparseHandle, &one, m_matA, m_vecp, &zero, m_vecAp, d_buffer);
			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_r, d_z);
			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_b, d_r);
			//CublasCaller<Scalar>::axpy(m_cublasHandle, m_n, &neg_one, d_Ap, d_r);
			//CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_r, d_r, &res);
			//hipMemcpy(test.data(), x, test.size() * sizeof(Scalar), hipMemcpyDeviceToHost);
			//for (int i = 0; i < m_n; ++i)
			//	std::cout << std::setprecision(std::numeric_limits<Scalar>::digits10 + 1) << test(i) << ' ';
			//std::cout << '\n';
			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_Ap, d_p);
			//CublasCaller<Scalar>::copy(m_cublasHandle, m_n, d_z, d_r);

			//if (res < eps * bnorm) break;

			m_precond->solve(d_r, d_z);
			CublasCaller<Scalar>::dot(m_cublasHandle, m_n, d_r, d_z, &rz);
			beta = rz / old_rz;
			CublasCaller<Scalar>::scal(m_cublasHandle, m_n, &beta, d_p);
			CublasCaller<Scalar>::axpy(m_cublasHandle, m_n, &one, d_z, d_p);
		}

		//std::cout << "Total CG iteration: " << k << " residual: " << res << std::endl;

		return true;
	}

	void DummyPreconditioner::solve(const Scalar* in, Scalar* out)
	{
		hipMemcpy(out, in, m_n * sizeof(Scalar), hipMemcpyDeviceToDevice);
	}

	DiagnalPreconditioner::DiagnalPreconditioner(SparseMatrix* A) :m_A(A)
	{
		checkCudaErrors(hipMalloc((void**)&m_invDiagA, A->getn() * sizeof(Scalar)));
	}

	DiagnalPreconditioner::~DiagnalPreconditioner()
	{
		hipFree(m_invDiagA);
	}

	bool DiagnalPreconditioner::analysis()
	{
		m_A->invDiagonal(m_invDiagA);
		return true;
	}

	__global__ void cwiseMultiply(int n, const Scalar* a, const Scalar* b, Scalar* c)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= n) return;

		c[i] = a[i] * b[i];
	}

	void DiagnalPreconditioner::solve(const Scalar* in, Scalar* out)
	{
		cwiseMultiply <<< get_block_num(m_A->getn()), g_block_dim >>> (m_A->getn(), m_invDiagA, in, out);
	}

	FactorizationPreconditioner::FactorizationPreconditioner(SparseMatrix* A) :m_A(A)
	{
		checkCudaErrors(hipsparseCreate(&m_cusparseHandle));

		checkCudaErrors(hipsparseCreateMatDescr(&m_descrA));
		checkCudaErrors(hipsparseSetMatType(m_descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(m_descrA, HIPSPARSE_INDEX_BASE_ZERO));

		// Creates ILU info and triangular solve info
		checkCudaErrors(hipsparseCreateCsrilu02Info(&m_infoILU));

		checkCudaErrors(hipsparseCreateCsrsv2Info(&m_infoL));
		checkCudaErrors(hipsparseCreateCsrsv2Info(&m_infoU));

		checkCudaErrors(hipsparseCreateMatDescr(&m_descrL));
		checkCudaErrors(hipsparseSetMatType(m_descrL, HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(m_descrL, HIPSPARSE_INDEX_BASE_ZERO));
		checkCudaErrors(hipsparseSetMatFillMode(m_descrL, HIPSPARSE_FILL_MODE_LOWER));
		checkCudaErrors(hipsparseSetMatDiagType(m_descrL, HIPSPARSE_DIAG_TYPE_UNIT));

		checkCudaErrors(hipsparseCreateMatDescr(&m_descrU));
		checkCudaErrors(hipsparseSetMatType(m_descrU, HIPSPARSE_MATRIX_TYPE_GENERAL));
		checkCudaErrors(hipsparseSetMatIndexBase(m_descrU, HIPSPARSE_INDEX_BASE_ZERO));
		checkCudaErrors(hipsparseSetMatFillMode(m_descrU, HIPSPARSE_FILL_MODE_UPPER));
		checkCudaErrors(hipsparseSetMatDiagType(m_descrU, HIPSPARSE_DIAG_TYPE_NON_UNIT));

		hipMalloc((void**)&m_valsILU, A->getnnz() * sizeof(Scalar));
		hipMalloc((void**)&m_y, A->getn() * sizeof(Scalar));

		size_t bufferSize = 0;
		int tmp;

		CusparseCaller<Scalar>::ilu_bufferSize(m_cusparseHandle, A->getn(), A->getnnz(), m_descrA,
			A->getValue(), A->getRowPtr(), A->getColIdx(), m_infoILU, &tmp);
		if (tmp > bufferSize) bufferSize = tmp;

		CusparseCaller<Scalar>::sv2_bufferSize(m_cusparseHandle, A->getn(), A->getnnz(), m_descrL,
			A->getValue(), A->getRowPtr(), A->getColIdx(), m_infoL, &tmp);
		if (tmp > bufferSize) bufferSize = tmp;

		CusparseCaller<Scalar>::sv2_bufferSize(m_cusparseHandle, A->getn(), A->getnnz(), m_descrU,
			A->getValue(), A->getRowPtr(), A->getColIdx(), m_infoU, &tmp);
		if (tmp > bufferSize) bufferSize = tmp;

		checkCudaErrors(hipMalloc(&m_buffer, bufferSize));
	}

	FactorizationPreconditioner::~FactorizationPreconditioner()
	{
		hipsparseDestroy(m_cusparseHandle);
		hipsparseDestroyMatDescr(m_descrA);

		hipsparseDestroyCsrilu02Info(m_infoILU);
		hipsparseDestroyCsrsv2Info(m_infoL);
		hipsparseDestroyCsrsv2Info(m_infoU);
		hipsparseDestroyMatDescr(m_descrL);
		hipsparseDestroyMatDescr(m_descrU);

		hipFree(m_valsILU);
		hipFree(m_y);
		hipFree(m_buffer);
	}

	bool FactorizationPreconditioner::analysis()
	{
		int n = m_A->getn(), nnz = m_A->getnnz();
		Scalar* values = m_A->getValue();
		const int* rowPtr = m_A->getRowPtr();
		const int* colIdx = m_A->getColIdx();

		int structural_zero, numerical_zero;

		// Perform analysis for ILU
		CusparseCaller<Scalar>::ilu_analysis(m_cusparseHandle, n, nnz, m_descrA, values, rowPtr, colIdx, m_infoILU, m_buffer);

		auto status = hipsparseXcsrilu02_zeroPivot(m_cusparseHandle, m_infoILU, &structural_zero);
		if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
			printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
		}

		hipMemcpy(m_valsILU, values, nnz * sizeof(Scalar), hipMemcpyDeviceToDevice);
		CusparseCaller<Scalar>::ilu(m_cusparseHandle, n, nnz, m_descrA, m_valsILU, rowPtr, colIdx, m_infoILU, m_buffer);

		status = hipsparseXcsrilu02_zeroPivot(m_cusparseHandle, m_infoILU, &numerical_zero);
		if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
			printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
		}

		// Perform analysis for trianguler
		CusparseCaller<Scalar>::sv2_analysis(m_cusparseHandle, n, nnz, m_descrL, m_valsILU, rowPtr, colIdx, m_infoL, m_buffer);
		CusparseCaller<Scalar>::sv2_analysis(m_cusparseHandle, n, nnz, m_descrU, m_valsILU, rowPtr, colIdx, m_infoU, m_buffer);

		return structural_zero < 0 && numerical_zero < 0;
	}

	void FactorizationPreconditioner::solve(const Scalar* in, Scalar* out)
	{
		Scalar one = 1.0;

		// out = U^-1 * L^-1 * in
		CusparseCaller<Scalar>::sv2_solve(m_cusparseHandle, m_A->getn(), m_A->getnnz(), &one, m_descrL,
			m_valsILU, m_A->getRowPtr(), m_A->getColIdx(), m_infoL, in, m_y, m_buffer);
		CusparseCaller<Scalar>::sv2_solve(m_cusparseHandle, m_A->getn(), m_A->getnnz(), &one, m_descrU,
			m_valsILU, m_A->getRowPtr(), m_A->getColIdx(), m_infoU, m_y, out, m_buffer);
	}
}